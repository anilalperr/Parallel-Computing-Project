#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <netcdf.h>

/* Handle errors by printing an error message and exiting with a
  * non-zero status. */
#define ERRCODE 2
#define ERR(e) {printf("Error: %s\n", nc_strerror(e)); exit(ERRCODE);}
#define ICAR_LATITUDE 224
#define ICAR_LONGITUDE 464
#define OBS_LATITUDE 222
#define OBS_LONGITUDE 462
#define OBS_TIME_DIM 11323
#define ICAR_TIME_DIM 11322
#define NUM_YEARS 31
#define NUM_GRIDS 64
#define ICAR_DIR_1980 "/glade/p/ral/hap/trude/conus_icar/orig_and_removed_biasc_icar_data/merged_era/merged_era_hist_1980.nc"
#define OBS_DIR_1980  "/glade/p/ral/hap/common_data/Maurer_met_full/pr/nldas_met_update.obs.daily.pr.1980.nc"

// create a struct to store dataset info
typedef struct dataset_info {
    int time_dim;
    int ncids[NUM_YEARS];
    int varids[NUM_YEARS];
    int year_lengths[NUM_YEARS];
    int lat;
    int lon;
} dataset_info_t;

/*
given a start directory returns the dimensions of the whole dataset with precipitation values from 1980 to 2010
preconditions: takes in a directory (string)
postconditions: returns a struct that includes all the necessary information about the dataset such as dimensions, ncids, and varids
*/
dataset_info_t read_dataset(const char* dir) {
    // find the length of the directory
    int dir_length = strlen(dir);

    // find the first half of this directory
    char* first_half = (char*) malloc((dir_length-6) * sizeof(char));
    memcpy(first_half, dir, dir_length-7);

    // variable for error checking
    int retval;

    // create a dataset_info
    dataset_info_t d_info;

    // create an iteration variable
    int year;

    for (year = 1980; year < 2011; ++year) {
        // crate a variable for the dataset ids and variable ids
        int ncid;
        int varid;

        // allocate a memory for the current directory
        char* cur_dir = (char*) malloc((dir_length+1) * sizeof(char));

        // copy the first half to the memory for the current directory
        strcpy(cur_dir, first_half);

        // concatenate the current year to the current_directory
        char str_year[5];
        snprintf(str_year, 10, "%d", year);
        strcat(cur_dir, str_year);

        // concatenate the netcdf extension
        strcat(cur_dir, ".nc");

        // import the first dataset
        if ((retval = nc_open(cur_dir, NC_NOWRITE, &ncid))) {
           ERR(retval);
        }

        // access the number of dimensions in the dataset
        int num_dim;
        if ((retval = nc_inq(ncid, &num_dim, NULL, NULL, NULL))) {
            ERR(retval);
        }

        // create an iteration variable
        int dim_id_num;

        // find the length of each dimension and update the info struct
        for (dim_id_num = 0; dim_id_num < num_dim; ++dim_id_num) {
            size_t len_dim;
            char* name_dim = (char*) malloc(sizeof(char)*(NC_MAX_NAME+1));

            if ((retval = nc_inq_dim(ncid, dim_id_num, name_dim, &len_dim))) {
                ERR(retval);
            }

            if (strcmp(name_dim, "time") == 0) {
                if (year == 1980) {
                    d_info.time_dim = (int) len_dim;
                }
                else {
                    d_info.time_dim += (int) len_dim;
                }

                d_info.year_lengths[year-1980] = (int) len_dim;
            }
            else if ((strcmp(name_dim, "latitude") == 0) || (strcmp(name_dim, "lat") == 0)){
                d_info.lat = (int) len_dim;
            }
            else if ((strcmp(name_dim, "longitude") == 0) || (strcmp(name_dim, "lon") == 0)){
                d_info.lon = (int) len_dim;
            }

            free(name_dim);
        }

        // find the id for the precipitation variable
        if ((retval = nc_inq_varid(ncid, "pr", &varid))) {
            if ((retval = nc_inq_varid(ncid, "icar_pcp", &varid)))
               ERR(retval);
        }

        // add the ncids and varids to the info card
        d_info.ncids[year-1980] = ncid;
        d_info.varids[year-1980] = varid;

        // free the memory reserved for the current directory
        free(cur_dir);
    }

    // now that we are done with collecting information free the first_half of current directory
    free(first_half);

    return d_info;
}


// given the info about different datasets, copies the data in netcdf to arr
// preconditions: takes in an array of (any time size, 224 latitude, 464 longitude values), an array of dataset ids
// an array of variable ids, and a time dimension which should match up with the length of datasets id array
// postconditions: does not return anything, copies the contents of netcdf files into the input array
void copy_data_icar(float arr[][ICAR_LATITUDE][ICAR_LONGITUDE], int ncids[], int varids[], int year_lengths[]) {
    // create a variable for error check in
    int retval;

    // create the iteration variable
    int year_num;
    int cur_year = 0;
    int time_dim = NUM_YEARS;

    // iterate over each year and read it into the array
    for (year_num = 0; year_num < time_dim; ++year_num) {
        // copy the data into this array
        if ((retval = nc_get_var_float(ncids[year_num], varids[year_num], &arr[cur_year][0][0])))
           ERR(retval);

        cur_year += year_lengths[year_num];
    }
}

// given the info about different datasets, copies the data in netcdf to arr
// preconditions: takes in an array of (any time size, 224 latitude, 464 longitude values), an array of dataset ids
// an array of variable ids, and a time dimension which should match up with the length of datasets id array
// postconditions: does not return anything, copies the contents of netcdf files into the input array
void copy_data_obs(float arr[][OBS_LATITUDE][OBS_LONGITUDE], int ncids[], int varids[], int year_lengths[]) {
    // create a variable for error check in
    int retval;

    // create the iteration variable
    int year_num;
    int cur_year = 0;
    int time_dim = NUM_YEARS;

    // iterate over each year and read it into the array
    for (year_num = 0; year_num < time_dim; ++year_num) {
        // copy the data into this array
        if ((retval = nc_get_var_float(ncids[year_num], varids[year_num], &arr[cur_year][0][0])))
           ERR(retval);

        cur_year += year_lengths[year_num];
    }
}

// find the day with the minimum difference to given day
// preconditions: takes in an array of differences, the day we are comparing, and the size of the time axis
// postconditions: returns an integer, the index that allows us to find the minimum difference
__device__ int find_min_index(float mean_difs[], int day, int total_day) {
    // create a variable for minimum index
    int min_index;

    // pick the starting day
    if (day != 0) {
        min_index = 0;
    }
    else {
        min_index = 1;
    }

    // create an iteration variable
    int d;
    // iterate over all the mean differences
    for (d = 0; d < total_day; d++) {
        // if we are not on the day being compared and the current min difs is less than the prev min dif
        if ((day != d) && (mean_difs[d] < mean_difs[min_index])) {
            // update min index
            min_index = d;
        }
    }
    // return the minimum index
    return min_index;
}

// store all the days with closest precipitation rates in an array
// preconditions: takes in full icar data, a closest index array to update, and total time dimension
// postconditions: does not return anything, updates the integer array named dif_nw with indices
__global__ void find_difs(float* icar_data, int* dif_nw) {
    // set the iteration variables
    int cur_left_lon = (threadIdx.x % 8) * 10;
    int cur_lower_lat = (threadIdx.x / 8) * 10 + 120;

    // find the other edges of this grid
    int cur_right_lon = cur_left_lon + 10;
    int cur_upper_lat = cur_lower_lat + 10;

    // find the start time and finish time
    int day = blockIdx.x;
     
    // inner loop iteration variables
    int lat;
    int lon;
    int comp_day;

    float mean_difs_for_this_grid[ICAR_TIME_DIM];
    for (comp_day = 0; comp_day < ICAR_TIME_DIM; ++comp_day)  {
        // store the differences for this grid and day in an array
        float sum_difs = 0;
        for (lat = cur_lower_lat; lat < cur_upper_lat; lat++) {
            for (lon = cur_left_lon; lon < cur_right_lon; lon++) {
                float day_val = *(icar_data + (day * (ICAR_LATITUDE * ICAR_LONGITUDE) + lat * ICAR_LONGITUDE + lon));
                float comp_val = *(icar_data + (comp_day * (ICAR_LATITUDE * ICAR_LONGITUDE) + lat * ICAR_LONGITUDE + lon));
                // calculate the mean square difference
                float dif = (day_val - comp_val) * (day_val - comp_val);
                // add it to the sum
                sum_difs += dif;
            }
        }

        // take the mean difference of this grid
        mean_difs_for_this_grid[comp_day] = sum_difs / 100;
     }
     // find the corresponding day that is closest to this day
     int min_index = find_min_index(mean_difs_for_this_grid, day, comp_day);

     int* cur_pointer = dif_nw + (threadIdx.x * ICAR_TIME_DIM + day);

     *cur_pointer = min_index;
}

// finds all the corresponding observations in the northwest region and updates the float array named cor_obs
// preconditions: takes in the corresponding observation array to update, corresponding index array, full observation dataset,
// total number of grids, the total days we have
// postconditions: does not return anything, updates the cor_obs array and bias corrects the north west region in the full_icar dataset
void gen_corresponding_obs(float cor_obs[][80][80], int dif_nw [][ICAR_TIME_DIM], float full_obs[][OBS_LATITUDE][OBS_LONGITUDE], int num_grids, int time_dim) { 
    // iterate over each grid an time
    for (int g = 0; g < num_grids; ++g) {
        // access the coordinates of this grid in cor_obs
        int low_lat_icar = 10 * (g / 8);
        int up_lat_icar = low_lat_icar + 10;

        int left_lon_icar = 10 * (g % 8);
        int right_lon_icar = left_lon_icar + 10;

        // iterate over time;
        for (int time = 0; time < time_dim; ++time) {
            int time_index = dif_nw[g][time];

            // iterate over the coordinates
            for (int lat = low_lat_icar; lat < up_lat_icar; lat++) {
                for (int lon = left_lon_icar; lon < right_lon_icar; lon++) {
                    // pad the zeros
                    if (lon < 2) {
                        cor_obs[time][lat][lon] = 0;
                    }
                    else {
                        // assign the corresponding observation to this coordinate
                        cor_obs[time][lat][lon] = full_obs[time_index+1][119 + lat][lon-2];
                    }
                }
            }
        }
    }
}

int main(int argc, char** argv) {
    // Allocate a space for the dataset id and variable id
    int* ncid_try = (int*) malloc(sizeof(int));
    int* varid_try = (int*) malloc(sizeof(int));
    int retval;
 
    // Access the dimensions and ids of the 1980 icar dataset
    dataset_info_t icar_info = read_dataset(ICAR_DIR_1980);

    printf("Icar Dimensions: %d, %d, %d\n", icar_info.time_dim, icar_info.lat, icar_info.lon);

    float full_icar[ICAR_TIME_DIM][ICAR_LATITUDE][ICAR_LONGITUDE];

    // copy the icar precipitation rate to this three dimensional array
    copy_data_icar(full_icar, icar_info.ncids, icar_info.varids, icar_info.year_lengths);

    float* p_icar = &full_icar[0][0][0];

    // create an array to store the days with closest precipiation rates
    int dif_nw[NUM_GRIDS][ICAR_TIME_DIM];

    int* p_dif = &dif_nw[0][0];

    // Access the dimensions and ids of the 1980 icar dataset
    dataset_info_t obs_info = read_dataset(OBS_DIR_1980);

    printf("Observation Dimensions: %d, %d, %d\n", obs_info.time_dim, obs_info.lat, obs_info.lon);

    // reserve a space in the memory for the new dataset
    float full_obs[OBS_TIME_DIM][OBS_LATITUDE][OBS_LONGITUDE];

    // copy the icar precipitation rate to this three dimensional array
    copy_data_obs(full_obs, obs_info.ncids, obs_info.varids, obs_info.year_lengths);

    float* p_icar_dev;

    // Allocate a space for the full icar data on GPU
    if (hipMalloc(&p_icar_dev, icar_info.time_dim * icar_info.lat * icar_info.lon * sizeof(float)) != hipSuccess) {
      fprintf(stderr, "Failed to allocate the full icar dataset on GPU\n");
      exit(2);
    }

    // Copy the cpu's full_icar array to the gpu with hipMemcpy
    if(hipMemcpy(p_icar_dev, p_icar, icar_info.time_dim * icar_info.lat * icar_info.lon * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
      fprintf(stderr, "Failed to copy the full icar dataset to GPU\n");
    }

    int* p_dif_dev;

    // Allocate a space for the difference of northwest values on GPU
    if (hipMalloc(&p_dif_dev, NUM_GRIDS * ICAR_TIME_DIM * sizeof(int)) != hipSuccess) {
      fprintf(stderr, "Failed to allocate the dif index array on GPU\n");
      exit(2);
    }

 
    // call the kernel function to find the differences
    find_difs<<<ICAR_TIME_DIM, NUM_GRIDS>>>(p_icar_dev, p_dif_dev);

    // Wait for the kernel to finish
    if(hipDeviceSynchronize() != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(hipPeekAtLastError()));
    }

    // Copy the result back from gpu to cpu
    if(hipMemcpy(p_dif, p_dif_dev, NUM_GRIDS * ICAR_TIME_DIM * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {
      fprintf(stderr, "Failed to copy result from the GPU\n");
    } 

    // free the memory
    hipFree(p_dif_dev); 
    hipFree(p_icar_dev);

    // create an array to store the bias corrected output
    float nw_cor_obs[icar_info.time_dim][80][80];

    // find the corresponding observations and assign it to the bias corrected dataset
    gen_corresponding_obs(nw_cor_obs, dif_nw, full_obs, 64, icar_info.time_dim);

    // print some numbers in this dataset
    printf("%f\n", nw_cor_obs[0][40][20]);
    printf("%f\n", nw_cor_obs[2][62][15]);
    printf("%f\n", nw_cor_obs[0][50][20]);
    printf("%f\n", nw_cor_obs[0][62][15]);
    printf("%f\n", nw_cor_obs[0][32][33]);

    //create the netcdf file
    // Allocate space for netCDF dimension ids
    int timeId, latId, lonId;

    // Allocate space for the netcdf file id
    int ncid;

    // Allocate space for the data variable ids
    int pcp_id;

    // array to store the dimension ids
    int dimids[3];

    // create a netcdf file named "updated_C_data.nc"
    if((retval = nc_create("updated_C_data.nc", NC_NETCDF4, &ncid))) {
        ERR(retval);
    }

    // define the dimensions (time, latitude, and longitude)
    if((retval = nc_def_dim(ncid, "time", icar_info.time_dim, &timeId))) {
        ERR(retval);
    }

    if((retval = nc_def_dim(ncid, "latitude", 80, &latId))) {
        ERR(retval);
    }

    if((retval = nc_def_dim(ncid, "longitude", 80, &lonId))) {
        ERR(retval);
    }

    dimids[0] = timeId;
    dimids[1] = latId;
    dimids[2] = lonId;

    // Add the variable
    if((retval = nc_def_var(ncid, "pcp", NC_FLOAT, 3, dimids, &pcp_id))) {
        ERR(retval);
    }

    // End "Metadata" mode
    if((retval = nc_enddef(ncid))) {
        ERR(retval);
    }

    // Add the bias-corrected data to this netcdf file
    if((retval = nc_put_var(ncid, pcp_id, &nw_cor_obs[0][0][0]))) {
        ERR(retval);
    }

    // Close the net_cdf file for bias-corrected-data
    if((retval = nc_close(ncid))) {
        ERR(retval);
    }
    return 0;
}
